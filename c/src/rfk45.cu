#include "hip/hip_runtime.h"
#define A2 (1/4)
#define B2 (1/4)

#define A3 (3/8)
#define B3 (3/32)
#define C3 (9/32)

#define	A4 (12/13)
#define B4 (1932/2197)
#define C4 (-7200/2197)
#define D4 (7296/2197)

#define A5 1
#define B5 (439/216)
#define C5 (-8)
#define D5 (3680/513)
#define E5 (-845/4104)

#define A6 (1/2)
#define B6 (-8/27)
#define C6 2
#define D6 (-3544/2565)
#define E6 (1859/4104)
#define F6 (-11/40)

#define R1 (1/360)
#define R3 (-128/4275)
#define R4 (-2197/75240)
#define R5 (1/50)
#define R6 (2/55)

#define N1 (25/216)
#define N3 (1408/2565)
#define N4 (2197/4104)
#define N5 (-1/5)

#define MINIMUM_TIME_STEP 0.0000

#ifndef KERNEL_RETURN_CODES
#define KERNEL_RETURN_CODES
#define CODE_TIMEOUT 1
#define CODE_SUCCESS 2
#define CODE_PRECISION_FAILED 3
#endif

typedef void (*t_ode_function) (
	const float, 	// time
	const float, 	// value of variable
	const int,	// index of given variable
	float*		// computed value
);

inline void __device__ example_f(const float time, const float value, const int index, float* result) {
	*result = value;
}

void __device__ rfk45_step(
	float time,
	const float 	expected_time_step,
	const float 	value,
	const int 	variable_index,
	const float 	abs_divergency,
	const float 	rel_divergency,
	const 		t_ode_function ode_function,
	float* 		result,
	float* 		changed_time_step
) {
	float k1, k2, k3, k4, k5, k6;
	ode_function(time + expected_time_step, value, variable_index, &k1);
	ode_function(time + A2 * expected_time_step, value + B2 * k1, variable_index, &k2);
	ode_function(time + A3 * expected_time_step, value + B3 * k1 + C3 * k2, variable_index, &k3);
	ode_function(time + A4 * expected_time_step, value + B4 * k1 + C4 * k2 + D4 * k3, variable_index, &k4);
	ode_function(time + A5 * expected_time_step, value + B5 * k1 + C5 * k2 + D5 * k3 + E5 * k4, variable_index, &k5);
	ode_function(time + A6 * expected_time_step, value + B6 * k1 + C6 * k2 + D6 * k3 + E6 * k4 + F6 * k5, variable_index, &k6);

	float error 	= abs(R1 * k1 + R3 * k3 + R4 * k4 + R5 * k5 + R6 * k6);
	float y		= value + N1 * k1 + N3 * k3 + N4 * k4 + N5 * k5;
	
void __global__ rfk45_kernel(
	/* INPUT */
	const float*	init_vectors,
	const int	number_of_vectors,
	const int	size_of_vector,
	const float	init_time,
	const float 	target_time,
	const float	time_step,
	const int	max_number_of_steps,
	const float	abs_divergency,
	const float	rel_divergency,
	const t_ode_function ode_function,
	/* OUTPUT */
	float*		return_code,
	int*		number_of_successful_steps,
	float*		simulation

) {

	int id = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
	if (id >= number_of_vectors) return;
	const float* previous_vector = &init_vectors[id * number_of_vectors];
	unsigned int current_step = 0;
	
	for(int i=0; i<(ceil(target_time/time_step)); i++) {
		current_step++;
		if (current_step > max_number_of_steps) {
			break;
		}
		
	}
	
	number_of_successful_steps[id] = current_step > max_number_of_steps ? max_number_of_steps : current_step;
}
